#include "hip/hip_runtime.h"
#pragma once
#include "header.cuh"


int main()
{
	clock_t start;   //Timer
	long double diff;
	start = clock();

	MC_Migraiton();

	diff = (std::clock() - start) / (long double)CLOCKS_PER_SEC;
	cout << "computation time: " << diff << '\n';
	system("pause");



	return 0;
}